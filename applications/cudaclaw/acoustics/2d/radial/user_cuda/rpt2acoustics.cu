#include "../swirl_user.h"

#include <fclaw_base.h>  /* Needed for SC_MIN, SC_MAX */


__device__ void swirl_rpt2adv(int idir, int meqn, int mwaves, int maux,
                              double ql[], double qr[], 
                              double aux1[], double aux2[], double aux3[],
                              int imp, int pm, double asdq[],
                              double bmasdq[], double bpasdq[])
{
    int mq, kv;

    /* Use pm to determine if we should compute bmasdq or bpasdq */
    kv = 1-idir;
    for(mq = 0; mq < meqn; mq++)
    {
        if (pm == 0)
        {
            bmasdq[mq] = SC_MIN(aux2[imp*maux + kv], 0) * asdq[mq];                        
        }
        else
        {
            bpasdq[mq] = SC_MAX(aux3[imp*maux + kv], 0) * asdq[mq];                                
        }
    }
}

__device__ cudaclaw_cuda_rpt2_t swirl_rpt2 = swirl_rpt2adv;

void swirl_assign_rpt2(cudaclaw_cuda_rpt2_t *rpt2)
{
    hipError_t ce = hipMemcpyFromSymbol(rpt2, HIP_SYMBOL(swirl_rpt2), sizeof(cudaclaw_cuda_rpt2_t));
    if(ce != hipSuccess)
    {
        fclaw_global_essentialf("ERROR (swirl_rpt2adv): %s\n",hipGetErrorString(ce));
        exit(0);
    }    
}